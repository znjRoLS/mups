#include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <time.h>
# include <omp.h>

#include "common.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>

//#define N 1024*1024*12

#define BLOCK_SIZE 256

#define MAX_BLOCKS 65535

#define MAX_MEMORY BLOCK_SIZE * MAX_BLOCKS

double f(double x) {
	double pi = 3.141592653589793;
	double value;

	value = 50.0 / (pi * (2500.0 * x * x + 1.0));

	return value;
}

__device__ double fDev(double x) {
	double pi = 3.141592653589793;
	double value;

	value = 50.0 / (pi * (2500.0 * x * x + 1.0));

	return value;
}

int sequential(int argc, char *argv[], Result_Vect *result) {
	double a;
	double b;
	double error;
	int i;
	int n;
	double total_q, total_t, total_s;
	double wtime_q, wtime_t, wtime_s;
	double x;
	double h;

	printf("\n\nSEQUENTIAL\n");
	
	result->time = 0;

	if (argc != 4) {
		n = 10000000;
		a = 0.0;
		b = 10.0;
	}
	else {
		n = atoi(argv[1]);
		a = atoi(argv[2]);
		b = atoi(argv[3]);
	}

	printf("\n");
	printf("QUAD:\n");
	printf("  Estimate the integral of f(x) from A to B.\n");
	printf("  f(x) = 50 / ( pi * ( 2500 * x * x + 1 ) ).\n");
	printf("\n");
	printf("  A        = %f\n", a);
	printf("  B        = %f\n", b);
	printf("  N        = %d\n", n);


	// Quadratic rule  
	wtime_q = omp_get_wtime();

	total_q = 0.0;

	for (i = 0; i < n; i++)
	{
		x = ((double)(n - i - 1) * a + (double)(i)* b) / (double)(n - 1);
		total_q = total_q + f(x);
	}

	wtime_q = omp_get_wtime() - wtime_q;

	total_q = (b - a) * total_q / (double)n;

	result->time += wtime_q;
	result->value[0] = total_q;


	// Trapezoidal rule  
	h = (b - a) / n;

	wtime_t = omp_get_wtime();

	total_t = 0.0;

	for (i = 0; i < n; i++)
	{
		x = a + i * h;
		if (i > 0 && i < n - 1)
			total_t = total_t + f(x);
		else
			total_t = total_t + 0.5 * f(x);
	}

	total_t = h * total_t;

	wtime_t = omp_get_wtime() - wtime_t;


	result->time += wtime_t;
	result->value[1] = total_t;

	// Simpson 1/3 rule  

	h = (b - a) / n;

	wtime_s = omp_get_wtime();

	total_s = 0.0;

	for (i = 0; i < n; i++)
	{
		x = a + i * h;
		if (i == 0 || i == n - 1)
			total_s = total_s + f(x);
		else if (i % 2 == 1)
			total_s = total_s + 4 * f(x);
		else
			total_s = total_s + 2 * f(x);
	}

	total_s = h / 3 * total_s;

	wtime_s = omp_get_wtime() - wtime_s;

	result->time += wtime_s;
	result->value[2] = total_s;

	printf("\n");
	printf("  Estimate quadratic rule = %24.16f\n", total_q);
	printf("  Estimate trapezoidal rule = %24.16f\n", total_t);
	printf("  Estimate Simpson 1/3 rule = %24.16f\n", total_s);
	printf("  Time quadratic rule = %f\n", wtime_q);
	printf("  Time trapezoidal rule = %f\n", wtime_t);
	printf("  Time Simpson 1/3 rule = %f\n", wtime_s);
	printf("\n");
	printf("  Normal end of execution.\n");
	printf("\n");

	return 0;
}

/////////////////// parallel

// Simple reduction kernel
__global__ void reductionSumKernel(double* devA, double* blockResults, int n) {
	extern __shared__ double sharedData[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	// Load block in the shared memory
	if (i < n) sharedData[tid] = devA[i];
	else sharedData[tid] = 0;

	__syncthreads();

	// Do reduction in shared memory
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sharedData[tid] += sharedData[tid + s];
		}
		__syncthreads();
	}

	// Write result for this block to global memory 
	if (tid == 0) blockResults[blockIdx.x] = sharedData[0];
}

double sumReduction(double* devA, int n) {
	double gpuSum = 0;
	int numBlocks = 0;
	double *devBlockRes;
	
	// Run kernel several times until the work is done
	numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	//printf("this is numblocks %d\n", numBlocks);
	hipMalloc((void **)&devBlockRes, numBlocks * sizeof(double));

	reductionSumKernel <<< numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double) >>>(devA, devBlockRes, n);

	while (numBlocks > 1) {
		n = numBlocks;
		numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
		reductionSumKernel <<< numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double) >>>(devBlockRes, devBlockRes, n);
	}

	// Copy back the results
	hipMemcpy(&gpuSum, devBlockRes, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(devBlockRes);

	return gpuSum;
}

__global__ void compute_kernel_quad_big(double *devA, double a, double b, int n, int offset, int nSize) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
	while (i < nSize + offset) {
		double x = ((double)(n - i - 1) * a + (double)(i)* b) / (double)(n - 1);
		devA[i-offset] = fDev(x);
		i += blockDim.x * gridDim.x;
	}
}


__global__ void compute_kernel_trapezoidal_big(double *devA, double a, double b, int n, int offset, int nSize) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
	while (i < nSize + offset) {
		double h = (b - a) / n;

		double x = a + i * h;
		if (i > 0 && i < n - 1)
			devA[i - offset] = fDev(x);
		else
			devA[i - offset] = 0.5 * fDev(x);

		i += blockDim.x * gridDim.x;
	}
}

__global__ void compute_kernel_simpson_big(double *devA, double a, double b, int n, int offset, int nSize) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
	if (i < nSize + offset) {
		double h = (b - a) / n;

		double x = a + i * h;
		if (i == 0 || i == n - 1)
			devA[i-offset] = fDev(x);
		else if (i % 2 == 1)
			devA[i-offset] = 4 * fDev(x);
		else
			devA[i-offset] = 2 * fDev(x);

		i += blockDim.x * gridDim.x;
	}
}

//__global__ void compute_kernel_quad(double *devA, double a, double b, int n) {
//	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if (i < n) {
//		double x = ((double)(n - i - 1) * a + (double)(i)* b) / (double)(n - 1);
//		devA[i] = fDev(x);
//	}
//}

//__global__ void compute_kernel_trapezoidal(double *devA, double a, double b, int n) {
//	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if (i < n) {
//		double h = (b - a) / n;
//
//		double x = a + i * h;
//		if (i > 0 && i < n - 1)
//			devA[i] = fDev(x);
//		else
//			devA[i] = 0.5 * fDev(x);
//	}
//}

//double parallel_trapezoidal(double a, double b, int n) {
//	double *devA;
//
//	hipMalloc((void **)&devA, n * sizeof(double));
//	//hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
//
//	int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
//
//	compute_kernel_trapezoidal << < numBlocks, BLOCK_SIZE >> > (devA, a, b, n);
//	double total_q = sumReduction(devA, n);
//
//	hipFree(devA);
//
//	return total_q;
//}

//__global__ void compute_kernel_simpson(double *devA, double a, double b, int n) {
//	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if (i < n) {
//		double h = (b - a) / n;
//
//		double x = a + i * h;		
//		if (i == 0 || i == n - 1)
//			devA[i] = fDev(x);
//		else if (i % 2 == 1)
//			devA[i] = 4 * fDev(x);
//		else
//			devA[i] = 2 * fDev(x);
//	}
//}

//double parallel_simpson(double a, double b, int n) {
//	double *devA;
//
//	hipMalloc((void **)&devA, n * sizeof(double));
//	//hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
//
//	int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
//
//	compute_kernel_simpson << < numBlocks, BLOCK_SIZE >> > (devA, a, b, n);
//	double total_q = sumReduction(devA, n);
//
//	hipFree(devA);
//
//	return total_q;
//}


double parallel_compute(double a, double b, int n, void (*kernel)(double*,double,double,int,int,int)) {
	double *devA;
	double total_q = 0;

	for (int ni = 0; ni < n; ni += MAX_MEMORY) {
		int nSize = MAX_MEMORY;
		if (n - ni < MAX_MEMORY)
			nSize = n - ni;

		hipMalloc((void **)&devA, nSize * sizeof(double));
		if (hipSuccess != hipGetLastError()) {
			printf("couldnt allocate %d doubles\n", nSize);
			break;
		}

		int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
		if (numBlocks > MAX_BLOCKS)
			numBlocks = MAX_BLOCKS;

		kernel << < numBlocks, BLOCK_SIZE >> > (devA, a, b, n, ni, nSize);
		if (hipSuccess != hipGetLastError()) {
			printf("error computing\n");
			break;
		}

		total_q += sumReduction(devA, nSize);
		if (hipSuccess != hipGetLastError()) {
			printf("error reducting\n");
			break;
		}
		
		hipFree(devA);
	}

	return total_q;	
}

int parallel(int argc, char *argv[], Result_Vect *result) {
	double a;
	double b;
	double error;
	int i;
	int n;
	double total_q, total_t, total_s;
	double wtime_q, wtime_t, wtime_s;
	double x;
	double h;

	printf("\n\nPARALLEL\n");
	
	result->time = 0;

	if (argc != 4) {
		n = 10000000;
		a = 0.0;
		b = 10.0;
	}
	else {
		n = atoi(argv[1]);
		a = atoi(argv[2]);
		b = atoi(argv[3]);
	}

	printf("\n");
	printf("QUAD:\n");
	printf("  Estimate the integral of f(x) from A to B.\n");
	printf("  f(x) = 50 / ( pi * ( 2500 * x * x + 1 ) ).\n");
	printf("\n");
	printf("  A        = %f\n", a);
	printf("  B        = %f\n", b);
	printf("  N        = %d\n", n);


	// Quadratic rule  
	wtime_q = omp_get_wtime();

	/*total_q = 0.0;

	for (i = 0; i < n; i++)
	{
		x = ((double)(n - i - 1) * a + (double)(i)* b) / (double)(n - 1);
		total_q = total_q + f(x);
	}*/

	total_q = parallel_compute(a, b, n, compute_kernel_quad_big);

	total_q = (b - a) * total_q / (double)n;

	wtime_q = omp_get_wtime() - wtime_q;



	result->time += wtime_q;
	result->value[0] = total_q;

	// Trapezoidal rule  
	h = (b - a) / n;

	wtime_t = omp_get_wtime();

	/*total_t = 0.0;

	for (i = 0; i < n; i++)
	{
		x = a + i * h;
		if (i > 0 && i < n - 1)
			total_t = total_t + f(x);
		else
			total_t = total_t + 0.5 * f(x);
	}*/

	/*total_t = parallel_trapezoidal(a, b, n);*/
	total_t = parallel_compute(a, b, n, compute_kernel_trapezoidal_big);

	total_t = h * total_t;

	wtime_t = omp_get_wtime() - wtime_t;

	result->time += wtime_t;
	result->value[1] = total_t;

	// Simpson 1/3 rule  

	h = (b - a) / n;

	wtime_s = omp_get_wtime();
/*
	total_s = 0.0;

	for (i = 0; i < n; i++)
	{
		x = a + i * h;
		if (i == 0 || i == n - 1)
			total_s = total_s + f(x);
		else if (i % 2 == 1)
			total_s = total_s + 4 * f(x);
		else
			total_s = total_s + 2 * f(x);
	}*/

	total_s = parallel_compute(a, b, n, compute_kernel_simpson_big);
	/*total_s = parallel_simpson(a, b, n);*/

	total_s = h / 3 * total_s;

	wtime_s = omp_get_wtime() - wtime_s;

	result->time += wtime_s;
	result->value[2] = total_s;

	printf("\n");
	printf("  Estimate quadratic rule = %24.16f\n", total_q);
	printf("  Estimate trapezoidal rule = %24.16f\n", total_t);
	printf("  Estimate Simpson 1/3 rule = %24.16f\n", total_s);
	printf("  Time quadratic rule = %f\n", wtime_q);
	printf("  Time trapezoidal rule = %f\n", wtime_t);
	printf("  Time Simpson 1/3 rule = %f\n", wtime_s);
	printf("\n");
	printf("  Normal end of execution.\n");
	printf("\n");

	return 0;
}


int main(int argc, char *argv[]) {

	//double sequential_result, parallel_result, sequential_time, parallel_time;

	Result_Vect seq_result;
	Result_Vect par_result;

	seq_result.val_size = 3;
	seq_result.value = (double*)malloc(3 * sizeof(double));
	par_result.val_size = 3;
	par_result.value = (double*)malloc(3 * sizeof(double));


	/*for (int i = 1; ; i <<= 1) {
		double *nekid;
		hipMalloc((void **)&nekid, i * sizeof(double));
		if (hipSuccess != hipGetLastError()) {
			printf("couldnt allocate %d doubles\n", i);
			break;
		}
		else {
			printf("allocated %d doubles\n", i);
			hipFree(nekid);
		}

	}*/

	sequential(argc, argv, &seq_result);
	parallel(argc, argv, &par_result);

	compare_and_print_vect(seq_result, par_result, "Numeric integration");
}
