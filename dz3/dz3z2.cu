#include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <time.h>

#include "common.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>


double cpu_time(void)
{
	double value;

	value = (double)clock() / (double)CLOCKS_PER_SEC;

	return value;
}


int sequential(int argc, char *argv[], Result_Vect *result)
{
	int M;
	int N;

	double ctime;
	double ctime1;
	double ctime2;
	double diff;
	double epsilon;
	FILE *fp;
	int i;
	int iterations;
	int iterations_print;
	int j;
	double mean;
	char output_file[80];
	int success;

	double **u;
	double **w;

	
	printf("\n\nSEQUENTIAL\n");
	
	if (argc != 5) {
		printf("Wrong number of arguments!\n");
		return 1;
	}
	else {
		success = sscanf(argv[1], "%d", &M);
		success += sscanf(argv[2], "%d", &N);
		success += sscanf(argv[3], "%lf", &epsilon);
		success += sscanf(argv[4], "%s", output_file);

		if (success != 4) {
			printf("Wrong arguments!\n");
			return 2;
		}
	}

	printf("\n");
	printf("HEATED_PLATE\n");
	printf("  C version\n");
	printf("  A program to solve for the steady state temperature distribution\n");
	printf("  over a rectangular plate.\n");
	printf("\n");
	printf("  Spatial grid of %d by %d points.\n", M, N);
	printf("\n");
	printf("  The iteration will be repeated until the change is <= %f\n", epsilon);
	diff = epsilon;
	printf("\n");
	printf("  The steady state solution will be written to %s.\n", output_file);

	u = (double **)malloc(M * sizeof(double*));
	for (i = 0; i < M; i++)
		u[i] = (double *)malloc(N * sizeof(double));

	w = (double **)malloc(M * sizeof(double*));
	for (i = 0; i < M; i++)
		w[i] = (double *)malloc(N * sizeof(double));

	/*
	Set the boundary values, which don't change.
	*/
	for (i = 1; i < M - 1; i++)
	{
		w[i][0] = 100.0;
	}
	for (i = 1; i < M - 1; i++)
	{
		w[i][N - 1] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[M - 1][j] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[0][j] = 0.0;
	}
	/*
	Average the boundary values, to come up with a reasonable
	initial value for the interior.
	*/
	mean = 0.0;
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i][0];
	}
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i][N - 1];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[M - 1][j];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[0][j];
	}
	mean = mean / (double)(2 * M + 2 * N - 4);
	/*
	Initialize the interior solution to the mean value.
	*/
	for (i = 1; i < M - 1; i++)
	{
		for (j = 1; j < N - 1; j++)
		{
			w[i][j] = mean;
		}
	}
	/*
	iterate until the  new solution W differs from the old solution U
	by no more than EPSILON.
	*/
	iterations = 0;
	iterations_print = 1;
	printf("\n");
	printf(" Iteration  Change\n");
	printf("\n");
	ctime1 = cpu_time();

	while (epsilon <= diff)
	{
		/*
		Save the old solution in U.
		*/
		for (i = 0; i < M; i++)
		{
			for (j = 0; j < N; j++)
			{
				u[i][j] = w[i][j];
			}
		}
		/*
		Determine the new estimate of the solution at the interior points.
		The new solution W is the average of north, south, east and west neighbors.
		*/
		diff = 0.0;


		for (i = 1; i < M - 1; i++)
		{
			for (j = 1; j < N - 1; j++)
			{
				w[i][j] = (u[i - 1][j] + u[i + 1][j] + u[i][j - 1] + u[i][j + 1]) / 4.0;

				if (diff < fabs(w[i][j] - u[i][j]))
				{
					diff = fabs(w[i][j] - u[i][j]);
				}
			}
		}

		/*for (i = 0; i < 3; i++)
			for (j = 0; j < 3; j++)
				if (i < 2 && j < 2) {
					printf("seq - devvv? %d %d %llf - %llf\n", i, j, u[i][j], w[i][j]);
				}*/

		iterations++;
		if (iterations == iterations_print)
		{
			printf("  %8d  %f\n", iterations, diff);
			iterations_print = 2 * iterations_print;
		}

		/*if (iterations == 5) {
			diff = 0;
		}*/
	}
	ctime2 = cpu_time();
	ctime = ctime2 - ctime1;

	printf("\n");
	printf("  %8d  %f\n", iterations, diff);
	printf("\n");
	printf("  Error tolerance achieved.\n");
	printf("  CPU time = %f\n", ctime);
	/*
	Write the solution to the output file.
	*/
	fp = fopen(output_file, "w");

	fprintf(fp, "%d\n", M);
	fprintf(fp, "%d\n", N);

	result->val_size = M*N;
	result->value = (double*)malloc(M*N * sizeof(double));
	result->time = ctime;

	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			fprintf(fp, "%6.2f ", w[i][j]);
			result->value[i*N + j] = w[i][j];
		}
		fputc('\n', fp);
	}
	fclose(fp);

	printf("\n");
	printf("  Solution written to the output file %s\n", output_file);
	/*
	All done!
	*/
	printf("\n");
	printf("HEATED_PLATE:\n");
	printf("  Normal end of execution.\n");

	return 0;

}

////////////////////////// parallel

//__device__ static double atomicMax(double* address, double val)
//{
//	unsigned long long int* address_as_i = 
//		(unsigned long long int*)address;
//	unsigned long long int old = *address_as_i, assumed;
//	do {
//		assumed = old;
//		old = ::atomicCAS(address_as_i, assumed,
//			__double_as_longlong( (val > __longlong_as_double(assumed) ) ? val : __longlong_as_double(assumed)));
//	} while (assumed != old);
//	return __longlong_as_double(old);
//}

__device__ void atomicMax(double * const address, const double value)
{
	if (*address >= value)
	{
		return;
	}

	unsigned long long int * const address_as_i = (unsigned long long int *)address;
	unsigned long long int old = *address_as_i, assumed;

	do
	{
		assumed = old;
		if (__longlong_as_double(assumed) >= value)
		{
			break;
		}

		old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
	} while (assumed != old);
}

__global__ void heated_kernel(double *devA, double *devB, int N, int M, double *epsilon) {
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;


	//printf("nesto? %d %d\n", i ,j);
	if (i > 0 && j > 0 && i < M-1 && j < N-1) {
		devB[i*N + j] = (devA[(i - 1)*N + j] + devA[(i + 1)*N + j] + devA[i * N + j - 1] + devA[i*N + j + 1]) / 4.0;
		
		atomicMax(epsilon, devB[i*N + j] - devA[i*N + j]);
		atomicMax(epsilon, devA[i*N + j] - devB[i*N + j]);
	}

	/*if (i < 2 && j < 2) {
		printf("devvv? %d %d %llf - %llf\n", i, j, devA[i*N + j], devB[i*N + j]);
	}*/
}

__global__ void heated_kernel2(double *devA, double *devB, int N, int M, double epsilon, int *isBigger) {
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;


	//printf("nesto? %d %d\n", i ,j);
	if (i > 0 && j > 0 && i < M - 1 && j < N - 1) {
		devB[i*N + j] = (devA[(i - 1)*N + j] + devA[(i + 1)*N + j] + devA[i * N + j - 1] + devA[i*N + j + 1]) / 4.0;

		if (devB[i*N + j] - devA[i*N + j] > epsilon) {
			*isBigger = 1;
		}
		if (devA[i*N + j] - devB[i*N + j] > epsilon) {
			*isBigger = 1;
		}
	}

	/*if (i < 2 && j < 2) {
	printf("devvv? %d %d %llf - %llf\n", i, j, devA[i*N + j], devB[i*N + j]);
	}*/
}

#define USEPLATE 1

void parallel_heated_plate(double **u, double **w, int N, int M, double epsilon) {

	double diff = epsilon;
	int iterations = 0;
	int iterations_print = 1;
	int i, j;
	double zeroDouble = 0.0;
	int zeroInt = 0;
	int isDiff = 1;

	dim3 threadsPerBlock(32, 16);
	dim3 numBlocks((M - 2 +threadsPerBlock.x - 1) / threadsPerBlock.x, (N - 2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

	double *devU, *devW;
	double *epsilonCuda;
	int *isDiffCuda;

	hipMalloc((void **)&devU, N*M * sizeof(double));
	hipMalloc((void **)&devW, N*M * sizeof(double));
	if (hipSuccess != hipGetLastError()) {
		printf("errorr------------------------------------------\n");
	}
	hipMalloc((void **)&epsilonCuda, sizeof(double));
	hipMalloc((void **)&isDiffCuda, sizeof(int));

	if (hipSuccess != hipGetLastError()) {
		printf("errorr------------------------------------------\n");
	}

	for (int i = 0; i < M; i++) {
		hipMemcpy(devU + i*N, u[i], N * sizeof(double), hipMemcpyHostToDevice);

		if (hipSuccess != hipGetLastError()) {
			printf("errorr------------------------------------------\n");
		}
	}

	for (int i = 0; i < M; i++) {
		hipMemcpy(devW + i*N, w[i], N * sizeof(double), hipMemcpyHostToDevice);

		if (hipSuccess != hipGetLastError()) {
			printf("errorr------------------------------------------\n");
		}
	}

	//hipMemcpy(devU, u, N*M * sizeof(double), hipMemcpyHostToDevice);

	int cnt = 0;

#if USEPLATE == 1
	while (epsilon <= diff)
#endif	
#if USEPLATE == 2
	while (isDiff)
#endif	
	{
		cnt++;

		if (USEPLATE == 1) {
			hipMemcpy(epsilonCuda, &zeroDouble, sizeof(double), hipMemcpyHostToDevice);
		}
		else if (USEPLATE == 2) {
			hipMemcpy(isDiffCuda, &zeroInt, sizeof(int), hipMemcpyHostToDevice);
		}
		
		/*
		Determine the new estimate of the solution at the interior points.
		The new solution W is the average of north, south, east and west neighbors.
		*/
		if (cnt % 2) {
			/*printf("%d u w\n", cnt);*/
			if (USEPLATE == 1) {
				heated_kernel << < numBlocks, threadsPerBlock >> > (devU, devW, N, M, epsilonCuda);
			}
			else if (USEPLATE == 2) {
				heated_kernel2 << < numBlocks, threadsPerBlock >> > (devU, devW, N, M, epsilon, isDiffCuda);
			}
			
		}
		else {
			/*printf("%d w u\n", cnt);*/
			
			if (USEPLATE == 1) {
				heated_kernel << < numBlocks, threadsPerBlock >> > (devW, devU, N, M, epsilonCuda);
			}
			else if (USEPLATE == 2) {
				heated_kernel2 << < numBlocks, threadsPerBlock >> > (devW, devU, N, M, epsilon, isDiffCuda);
			}
			
		}

		hipDeviceSynchronize();
		if (hipSuccess != hipGetLastError()) {
			printf("errorr------------------------------------------\n");
		}

		if (USEPLATE == 1) {
			hipMemcpy(&diff, epsilonCuda, sizeof(double), hipMemcpyDeviceToHost);
		}
		else if (USEPLATE == 2) {
			hipMemcpy(&isDiff, isDiffCuda, sizeof(int), hipMemcpyDeviceToHost);
		}
		
		//if (hipSuccess != hipGetLastError()) {
		//	printf("errorr\n");
		//}

		iterations++;
		if (iterations == iterations_print)
		{
			printf("  %8d  %f\n", iterations, diff);
			iterations_print = 2 * iterations_print;
		}
		/*if (iterations == 5) {
			diff = 0;
		}*/
	}

	if (cnt % 2) {
		for (int i = 0; i < M; i++) {
			hipMemcpy(w[i], devW + i*N, N * sizeof(double), hipMemcpyDeviceToHost);
			if (hipSuccess != hipGetLastError()) {
				printf("errorr\n");
			}
			//printf("sta bre? %d, %llf, %llf, %llf\n", i, w[i][0], w[i][1], w[i][2]);
		}
	}
	else {
		for (int i = 0; i < M; i++) {
			hipMemcpy(w[i], devU + i*N, N * sizeof(double), hipMemcpyDeviceToHost);
			if (hipSuccess != hipGetLastError()) {
				printf("errorr\n");
			}
			//printf("sta bru? %d, %llf, %llf, %llf\n", i, w[i][0], w[i][1], w[i][2]);
		}
	}

	printf("\n");
	printf("  %8d  %f\n", iterations, diff);

}


int parallel(int argc, char *argv[], Result_Vect *result)
{
	int M;
	int N;

	double ctime;
	double ctime1;
	double ctime2;
	double diff;
	double epsilon;
	FILE *fp;
	int i;
	int iterations;
	int iterations_print;
	int j;
	double mean;
	char output_file[80];
	int success;

	double **u;
	double **w;

	
	printf("\n\nPARALLEL\n");
	
	if (argc != 5) {
		printf("Wrong number of arguments!\n");
		return 1;
	}
	else {
		success = sscanf(argv[1], "%d", &M);
		success += sscanf(argv[2], "%d", &N);
		success += sscanf(argv[3], "%lf", &epsilon);
		success += sscanf(argv[4], "%s", output_file);

		if (success != 4) {
			printf("Wrong arguments!\n");
			return 2;
		}
	}

	printf("\n");
	printf("HEATED_PLATE\n");
	printf("  C version\n");
	printf("  A program to solve for the steady state temperature distribution\n");
	printf("  over a rectangular plate.\n");
	printf("\n");
	printf("  Spatial grid of %d by %d points.\n", M, N);
	printf("\n");
	printf("  The iteration will be repeated until the change is <= %f\n", epsilon);
	diff = epsilon;
	printf("\n");
	printf("  The steady state solution will be written to %s.\n", output_file);

	u = (double **)malloc(M * sizeof(double*));
	for (i = 0; i < M; i++)
		u[i] = (double *)malloc(N * sizeof(double));

	w = (double **)malloc(M * sizeof(double*));
	for (i = 0; i < M; i++)
		w[i] = (double *)malloc(N * sizeof(double));

	/*
	Set the boundary values, which don't change.
	*/
	for (i = 1; i < M - 1; i++)
	{
		w[i][0] = 100.0;
	}
	for (i = 1; i < M - 1; i++)
	{
		w[i][N - 1] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[M - 1][j] = 100.0;
	}
	for (j = 0; j < N; j++)
	{
		w[0][j] = 0.0;
	}
	/*
	Average the boundary values, to come up with a reasonable
	initial value for the interior.
	*/
	mean = 0.0;
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i][0];
	}
	for (i = 1; i < M - 1; i++)
	{
		mean = mean + w[i][N - 1];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[M - 1][j];
	}
	for (j = 0; j < N; j++)
	{
		mean = mean + w[0][j];
	}
	mean = mean / (double)(2 * M + 2 * N - 4);
	/*
	Initialize the interior solution to the mean value.
	*/
	for (i = 1; i < M - 1; i++)
	{
		for (j = 1; j < N - 1; j++)
		{
			w[i][j] = mean;
		}
	}
	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			u[i][j] = w[i][j];
		}
	}

	/*
	iterate until the  new solution W differs from the old solution U
	by no more than EPSILON.
	*/
	iterations = 0;
	iterations_print = 1;
	printf("\n");
	printf(" Iteration  Change\n");
	printf("\n");
	ctime1 = cpu_time();

	parallel_heated_plate(u, w, N, M, epsilon);

	//while (epsilon <= diff)
	//{
	//	/*
	//	Save the old solution in U.
	//	*/
	//	for (i = 0; i < M; i++)
	//	{
	//		for (j = 0; j < N; j++)
	//		{
	//			u[i][j] = w[i][j];
	//		}
	//	}
	//	/*
	//	Determine the new estimate of the solution at the interior points.
	//	The new solution W is the average of north, south, east and west neighbors.
	//	*/
	//	diff = 0.0;
	//	for (i = 1; i < M - 1; i++)
	//	{
	//		for (j = 1; j < N - 1; j++)
	//		{
	//			w[i][j] = (u[i - 1][j] + u[i + 1][j] + u[i][j - 1] + u[i][j + 1]) / 4.0;

	//			if (diff < fabs(w[i][j] - u[i][j]))
	//			{
	//				diff = fabs(w[i][j] - u[i][j]);
	//			}
	//		}
	//	}
	//	iterations++;
	//	if (iterations == iterations_print)
	//	{
	//		printf("  %8d  %f\n", iterations, diff);
	//		iterations_print = 2 * iterations_print;
	//	}
	//}
	ctime2 = cpu_time();
	ctime = ctime2 - ctime1;

	printf("\n");
	printf("  Error tolerance achieved.\n");
	printf("  CPU time = %f\n", ctime);
	/*
	Write the solution to the output file.
	*/
	fp = fopen(output_file, "w");

	fprintf(fp, "%d\n", M);
	fprintf(fp, "%d\n", N);

	result->val_size = M*N;
	result->value = (double*)malloc(M*N * sizeof(double));
	result->time = ctime;

	for (i = 0; i < M; i++)
	{
		//printf("sta brej? %d, %llf, %llf, %llf\n", i, w[i][0], w[i][1], w[i][2]);

		for (j = 0; j < N; j++)
		{
			fprintf(fp, "%6.2f ", w[i][j]);
			result->value[i*N + j] = w[i][j];
		}
		fputc('\n', fp);
	}
	fclose(fp);

	printf("\n");
	printf("  Solution written to the output file %s\n", output_file);
	/*
	All done!
	*/
	printf("\n");
	printf("HEATED_PLATE:\n");
	printf("  Normal end of execution.\n");

	return 0;

}

int main(int argc, char * argv[]) {
	Result_Vect seq_result, par_result;

	sequential(argc, argv, &seq_result);
	parallel(argc, argv, &par_result);

	compare_and_print_vect(seq_result, par_result, "heated plate");
}